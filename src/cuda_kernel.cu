//nvcc -ptx cuda_kernel.cu --gpu-architecture=compute_30 --gpu-code=compute_30
//In case of compilation error: https://github.com/arrayfire/arrayfire/issues/1384
#include "hip/hip_complex.h"
// CUDA runtime
#include "hip/hip_runtime.h"
#include "stdint.h"

#define IDX2F(i,j,ld) ((((j)-1)*(ld))+((i)-1))
#define IDX2C(i,j,ld) (((j)*(ld))+(i))
// #define IDX3(k1,k2,k3,kcut) (((((k1)-1)*(kcut))+((k2)-1))*(kcut) + ((k3)-1))

extern "C"
{
  //Multiplies the coefficients according to indices list
  __global__ void calculate_coefficient_matrix(const hipFloatComplex* coeff, const int numcoeff, const float* wignerlist, const int* indices, const int indiceslength, const int* PAcombos, const int combolength, float *PA, const int klength)
  {
    int i = threadIdx.x + blockIdx.x * blockDim.x + 1;
    if(i<=combolength)
    {
      const int k1 = PAcombos[IDX2F(1, i, 9)];
      const int k2 = PAcombos[IDX2F(2, i, 9)];
      const int k3 = PAcombos[IDX2F(3, i, 9)];
      const int ki = PAcombos[IDX2F(4, i, 9)];
      const int jstart = PAcombos[IDX2F(8, i, 9)];
      const int mcombos = PAcombos[IDX2F(9, i, 9)];

      const hipFloatComplex* ck1 = &coeff[(k1-1)*numcoeff];
      const hipFloatComplex* ck2 = &coeff[(k2-1)*numcoeff];
      const hipFloatComplex* ck3 = &coeff[(k3-1)*numcoeff];

      float As = 0.0f;
      for(int n=0; n < mcombos; n++){
        const int j = jstart + n;
        const int k1i = indices[IDX2F(1, j, 9)];
        const int k2i = indices[IDX2F(2, j, 9)];
        const int k3i = indices[IDX2F(3, j, 9)];
        As += wignerlist[j-1]*hipCrealf(hipCmulf(ck1[k1i-1], hipCmulf(ck2[k2i-1],ck3[k3i-1])) );
      }

      for(int n=0; n <mcombos; n++){
        PA[IDX2F(jstart+n,ki, indiceslength)] *= As;
      }
    }
  }
}
